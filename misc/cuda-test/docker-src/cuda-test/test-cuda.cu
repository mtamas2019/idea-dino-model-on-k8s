#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    std::cout << "Device Name: " << prop.name << std::endl;
    std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
    std::cout << "Total Global Memory: " << prop.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Max Threads Per Block: " << prop.maxThreadsPerBlock << std::endl;

    // GPU Memory Test
    const int numElements = 1000000;
    const size_t size = numElements * sizeof(float);

    float* d_array;
    hipMalloc(&d_array, size);

    if (d_array == nullptr) {
        std::cout << "Failed to allocate GPU memory" << std::endl;
        return 1;
    }

    hipFree(d_array);

    return 0;
}
